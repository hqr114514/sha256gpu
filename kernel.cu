#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <bits/stdc++.h>
using namespace std;

using ui = unsigned int;
using ull = unsigned long long;
using ll = long long;
using uc = unsigned char;

#define S(x,y) (((x)>>(y))|(x)<<(32-(y)))
#define swap32(x) ((x >> 24) | ((x >> 8) & 0xff00) | ((x << 8) & 0xff0000) | (x << 24))
#define Ch(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define Ma(x,y,z) (((x)&(y))^((x)&(z))^((y)&(z)))
#define S0(x) (S((x),2)^S((x),13)^S((x),22))
#define S1(x) (S((x),6)^S((x),11)^S((x),25))
#define o0(x) (S((x),7)^S((x),18)^((x)>>3))
#define o1(x) (S((x),17)^S((x),19)^((x)>>10))

__device__ const ui k[] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

__device__ ui H[8] = {
	0x6a09e667,
	0xbb67ae85,
	0x3c6ef372,
	0xa54ff53a,
	0x510e527f,
	0x9b05688c,
	0x1f83d9ab,
	0x5be0cd19
};

__device__ uc* dat;
__device__ ui* w;

__global__ void block(ull num) {
	ull start = (blockIdx.x * blockDim.x + threadIdx.x);//block_number
	if (start >= num) return;
	//block_number
	printf("%d\n", start);
	//转换:
	printf("execute to 0\nand start*64=%d\n", start * 64);
	for (ull t = start * 64; t <= start * 64 + 64; t++) printf("%02x\n", w[t]);
	for (ull t = start * 64, i = 1, j = t; i <= 16; i++, t++, j += 4) {
		printf("t=%lld i=%lld j=%lld\n", t, i, j);
		ui abcd = (dat[j] << 24) /* | (dat[j + 1] << 16) | (dat[j + 2] << 8) | dat[j + 3]*/;
		printf("finish\n");
	}
	printf("execute to 1\n");
	//扩充:
	for (int t = start * 64 + 16; t < (start + 1) * 64; t++) w[t] = o1(w[t - 2]) + w[t - 7] + o0(w[t - 15]) + w[t - 16];
	printf("execute to 2\n");
	
	/*ui a = H[0];
	ui b = H[1];
	ui c = H[2];
	ui d = H[3];
	ui e = H[4];
	ui f = H[5];
	ui g = H[6];
	ui h = H[7];
	for (int i = 0; i < 64; i++) {
		ui T1 = h + S1(e) + Ch(e, f, g) + k[i] + w[i];
		ui T2 = S0(a) + Ma(a, b, c);
		h = g;
		g = f;
		f = e;
		e = d + T1;
		d = c;
		c = b;
		b = a;
		a = T1 + T2;
	}
	H[0] += a;
	H[1] += b;
	H[2] += c;
	H[3] += d;
	H[4] += e;
	H[5] += f;
	H[6] += g;
	H[7] += h;*/
	return;
}

void sha256(const char* FileName, uc* out) {
	ifstream fin(FileName, ios::binary);
	fin.seekg(0, ios::end);//文件指针移动至最后
	ull siz = fin.tellg();//获取总长度
	fin.seekg(0, ios::beg);//文件指针移到开头
	uc* data = (uc*)malloc((siz / 64 + 1) * 64);
	cout << "number=" << (siz / 64 + 1) * 64 << endl;
	hipMalloc(&dat, (siz / 64 + 1) * 64);//64byte
	hipMemset(dat, 0, (siz / 64 + 1) * 64);
	hipMalloc(&w, (siz / 64 + 1) * 256ull);
	hipMemset(w, 0, (siz / 64 + 1) * 256ull);
	memset(data, 0, (siz / 64 + 1) * 64);
	ull cnt = 0,b = siz / 64 + (siz % 64 != 0);//block数量
	while (fin.read((char*)data, 64) && ((siz % 64 == 0 && cnt <= b) || cnt < b)) {//循环处理512bit的块,将整个文件读入
		cnt++;
	}
	//处理剩余
	ui rest = fin.gcount();
	fin.close();
	data[cnt * 64 + rest] = 0x80;//填1
	//for (ull i = cnt * 64 + rest; i < siz; i++) data[i] = 0;//填0

	for (ull i = siz - 8, j = 0; i < siz; i++, j++) {//8byte(64bit) length information
		data[i] = (siz * 8) >> ((7 - j) * 8);
		//此公式不用考虑剩余空间不够的问题(直接在数组后面写)
	}
	hipMemcpyToSymbol(HIP_SYMBOL(dat), data, (siz / 64 + 1) * 64);
	block << <b / 1024ull + 1ull, 1024 >> > (b);
	hipDeviceSynchronize();
	ui h[8] = { 0 };
	hipMemcpyFromSymbol(h, HIP_SYMBOL(H), 8 * sizeof(ui));
	for (int i = 0, j = 0; i < 32; i++, j += 8) {
		if (j == 32) j = 0;
		out[i] = swap32(h[i / 4]) >> j;
	}
	cout << "size=" << siz << endl;
	return;
}

int main(int argc, char** argv) {
	if (argc < 2) return 1;
	char* FileName = argv[1];
	uc out[32] = { 0 };
	sha256(FileName, out);
	for (int i = 0; i < 32; i++) printf("%02x", out[i]);
	cout << endl;
	return 0;
}