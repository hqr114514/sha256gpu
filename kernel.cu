#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <bits/stdc++.h>
using namespace std;

using ui = unsigned int;
using ull = unsigned long long;
using ll = long long;
using uc = unsigned char;

#define S(x,y) (((x)>>(y))|(x)<<(32-(y)))
#define swap32(x) ((x >> 24) | ((x >> 8) & 0xff00) | ((x << 8) & 0xff0000) | (x << 24))
#define Ch(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define Ma(x,y,z) (((x)&(y))^((x)&(z))^((y)&(z)))
#define S0(x) (S((x),2)^S((x),13)^S((x),22))
#define S1(x) (S((x),6)^S((x),11)^S((x),25))
#define o0(x) (S((x),7)^S((x),18)^((x)>>3))
#define o1(x) (S((x),17)^S((x),19)^((x)>>10))

const ui k[] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

ui H[8] = {
	0x6a09e667,
	0xbb67ae85,
	0x3c6ef372,
	0xa54ff53a,
	0x510e527f,
	0x9b05688c,
	0x1f83d9ab,
	0x5be0cd19
};

__device__ uc* dat;
__device__ ui* w;

__global__ void block(ull num, uc* dat,  ui* w) {
	ull start = (blockIdx.x * blockDim.x + threadIdx.x);//block_number
	if (start >= num) return;
	//转换:
	for (ull t = start * 64, i = 1, j = t; i <= 16; i++, t++, j += 4) {
		w[t] = ((dat[j]) << 24) | (dat[j + 1] << 16) | (dat[j + 2] << 8) | dat[j + 3];
	}
	//扩充:
	for (ull t = start * 64 + 16; t < (start + 1) * 64; t++) w[t] = o1(w[t - 2]) + w[t - 7] + o0(w[t - 15]) + w[t - 16];
	return;
}

void sha256(const char* FileName, uc* out) {
	ifstream fin(FileName, ios::binary);
	fin.seekg(0, ios::end);//文件指针移动至最后
	ull siz = fin.tellg();//获取总长度
	fin.seekg(0, ios::beg);//文件指针移到开头
	uc* data = (uc*)malloc((siz / 64 + 1) * 64);
	ull b = siz / 64 + 1 + (siz % 64 > 55);//block数量
	hipMalloc(&dat, (siz / 64 + 1) * 64);
	hipMemset(dat, 0, (siz / 64 + 1) * 64);
	hipMalloc(&w, (siz / 64 + 1) * 256ull);
	hipMemset(w, 0, (siz / 64 + 1) * 256ull);
	memset(data, 0, (siz / 64 + 1) * 64);
	fin.read((char*)data, siz);
	//处理剩余
	ui rest = siz % 64;
	fin.close();
	data[(b - 1) * 64 + rest] = 0x80;//填1
	for (ull i = (siz / 64 + 1) * 64 - 8, j = 0; i < (siz / 64 + 1) * 64; i++, j++)//8byte(64bit) length information
		data[i] = (siz * 8) >> ((7 - j) * 8);
	hipMemcpy(dat, data, (siz / 64 + 1) * 64, hipMemcpyHostToDevice);
	block << <b / 1024ull + 1ull, 1024 >> > (b, dat, w);
	hipDeviceSynchronize();
	ui* ww = (ui*)malloc((siz / 64 + 1) * 256ull);
	hipMemcpy(ww, w, (siz / 64 + 1) * 256ull, hipMemcpyDeviceToHost);
	for (ull i = 0; i < b; i++) {//遍历每个块
		ui a = H[0],b = H[1],c = H[2],d = H[3],e = H[4],f = H[5],g = H[6],h = H[7];
		for (ull j = 0; j < 64; j++) {
			ui T1 = h + S1(e) + Ch(e, f, g) + k[j] + ww[i * 64 + j];
			ui T2 = S0(a) + Ma(a, b, c);
			h = g;
			g = f;
			f = e;
			e = d + T1;
			d = c;
			c = b;
			b = a;
			a = T1 + T2;
		}
		H[0] += a,H[1] += b,H[2] += c,H[3] += d,H[4] += e,H[5] += f,H[6] += g,H[7] += h;
	}
	for (int i = 0, j = 0; i < 32; i++, j += 8) {
		if (j == 32) j = 0;
		out[i] = swap32(H[i / 4]) >> j;
	}
	return;
}

int main(int argc, char** argv) {
	if (argc < 2) return 1;
	char* FileName = argv[1];
	uc out[32] = { 0 };
	sha256(FileName, out);
	for (int i = 0; i < 32; i++) printf("%02x", out[i]);
	cout << endl;
	return 0;
}